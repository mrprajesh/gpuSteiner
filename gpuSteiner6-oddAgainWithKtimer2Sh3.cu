#include "hip/hip_runtime.h"
/*
 * // For IJPP'22 paper at https://doi.org/10.1007/s10766-021-00723-0.
 *
 * // For compiling
 * nvcc gpuSteiner6-oddAgainWithKtimer2Sh3.cu -o gpuSteiner6-oddAgainWithKtimer2Sh3.out -Wno-deprecated-gpu-targets -std=c++11 
 *
 * // Authors
 * Rajesh Pandian M | https://mrprajesh.co.in
 * Rupesh Nasre     | www.cse.iitm.ac.in/~rupesh
 * N.S.Narayanaswamy| www.cse.iitm.ac.in/~swamy
 *
 * MIT LICENSE
 *
 */
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include <map>
#include <set>
#include <unordered_map>
#include <vector>

#include "CUDAMST.cu"
using namespace std;

#define MAX_INT_IN_SHARED_PER_BLOCK 12288
#define SH_REGS_PER_THREAD 24

#define cudaCheckError()                                                               \
  {                                                                                    \
    hipError_t e = hipGetLastError();                                                \
    if (e != hipSuccess) {                                                            \
      printf("Cuda failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
      exit(1);                                                                         \
    }                                                                                  \
  }
#define TRUE 1
#define FALSE 0

//~ #define MAX_THREADS_PER_BLOCK 1024 // It is defined in CUDAMST.h

#define MAX_COST 1073741823
#define LEVEL 0
// 0 submit level-- no print
// 1 debug level -- prints as needed.

#define DEBUG if (LEVEL)

hipEvent_t tstart, tstop;
float totalTimeMilliSec = 0.0;
int sCount = 2;  // DEFAULT

__global__ void cpyParentArrayNew(int N, int index, int* d_parentArrays, int* d_parent, int sCount, int tempScount) {
  unsigned id = blockIdx.x * blockDim.x + threadIdx.x;  // Changed
  if (id < tempScount * N) {
    d_parentArrays[sCount * N * index + id] = d_parent[id];  // next block after sCount*N many
  }
}

__global__ void kernelInitDistAndParent(int N, int* minDist, int* parent, int tempScount) {  // FOR KSSSP

  unsigned id = threadIdx.x + blockDim.x * blockIdx.x;
  if (id < tempScount * N) {
    minDist[id] = INT_MAX / 2;
    parent[id] = -1;
  }
}

__global__ void kernelInitSources(int N, int* source, int* minDist, int tempScount) {  // FOR KSSSP
  unsigned id = threadIdx.x + blockDim.x * blockIdx.x;
  if (id == 0) {
    for (int ii = 0; ii < tempScount; ++ii)
      minDist[N * ii + source[ii]] = 0;
  }
}

__global__ void csrKernelBellmanFordMoore(int N, int* source,  // K SSSP  PULLL//PULLL
                                          int* csrM, int* csrD, int* csrW,
                                          bool* changed,
                                          int* minDist, int* parent,
                                          int sCount,
                                          int tempScount) {
  unsigned id = threadIdx.x + blockDim.x * blockIdx.x;

  if (id < tempScount * N) {
    int u = id;             // may not be needed but easier to code! -- ok! .For reading output
    int uIn = id % N;       // for read from input
    int start = csrM[uIn];  // may not be needed but easier to code! --refereed once :P
    int end = csrM[uIn + 1];
    int i, v, old, newDist, minSize;

    // variables to implement ShMEM
    int size = end - start;  // adjList size per thread
    int j;
    __shared__ int shD[MAX_INT_IN_SHARED_PER_BLOCK];  // for now. 12288  min(12288, _2M) -- it can never spill over

    // Sh MEM: cp csD
    for (i = start, j = 0, minSize = (size < SH_REGS_PER_THREAD ? size : SH_REGS_PER_THREAD); i < end && j < minSize; ++i, ++j)
      shD[threadIdx.x * SH_REGS_PER_THREAD + j] = csrD[i];

    //******** 1 PULL SH MEM **********
    for (i = start, j = 0; i < end; i++, j++) {  // 1 PULL SH MEM
      if (j < SH_REGS_PER_THREAD)
        v = shD[threadIdx.x * SH_REGS_PER_THREAD + j];  // If in ShMem take it
      else
        v = csrD[i];  // Else Read from Global
      newDist = minDist[(id / N) * N + v] + csrW[i];
      old = minDist[u];
      if (newDist < old) {
        minDist[u] = newDist;
        parent[u] = v;
        changed[0] = 1;
      }
    }
    //******** 2 PULL SH MEM **********
    for (i = start, j = 0; i < end; i++, j++) {  // 1 PULL SH MEM
      if (j < SH_REGS_PER_THREAD)
        v = shD[threadIdx.x * SH_REGS_PER_THREAD + j];  // If in ShMem take it
      else
        v = csrD[i];  // Else Read from Global
      newDist = minDist[(id / N) * N + v] + csrW[i];
      old = minDist[u];
      if (newDist < old) {
        minDist[u] = newDist;
        parent[u] = v;
        changed[0] = 1;
      }
    }
    //******** 3 PULL SH MEM **********
    for (i = start, j = 0; i < end; i++, j++) {  // 1 PULL SH MEM
      if (j < SH_REGS_PER_THREAD)
        v = shD[threadIdx.x * SH_REGS_PER_THREAD + j];  // If in ShMem take it
      else
        v = csrD[i];  // Else Read from Global
      newDist = minDist[(id / N) * N + v] + csrW[i];
      old = minDist[u];
      if (newDist < old) {
        minDist[u] = newDist;
        parent[u] = v;
        changed[0] = 1;
      }
    }
  }
}

void PrintParentOf(int n, int* pArray, int shift, int u, int v, set<pair<int, int>>& stEdges) {
  int idx = v;
  while (pArray[n * shift + idx] != -1) {
    int oldIdx = idx;

    idx = pArray[n * shift + idx];

    if (oldIdx < idx) {
      stEdges.insert(make_pair(oldIdx, idx));
    } else {
      stEdges.insert(make_pair(idx, oldIdx));
    }
  }
}

void KMBAlgo(int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
bool printEdges = false;
bool printHash = false;

int main(int argc, char** argv) {
  if (argc == 1) {
    printf("Usage: %s n -p\nn: #SSSPs in parallel. Default n=2\n", argv[0]);
    exit(0);
  }
  if (argc > 1) {
    //~ printEdges=true; //prints edges
    sCount = ((atoi(argv[1]) == 0) ? 2 : atoi(argv[1]));
    printHash = true;
  }
  if (argc > 2) {
    printEdges = true;
  }

  no_of_nodes = 0;
  edge_list_size = 0;
  KMBAlgo(argc, argv);

  return EXIT_SUCCESS;
}

int *edges, *edges_wt;
int* h_parentArrays;
int* d_parentArrays;
int N;
set<pair<int, int>> eSet;
set<int> vSet;

void MSTGraph(int t, int* terminals, map<pair<int, int>, int>& W, set<pair<int, int>>& stEdges, set<int>& nodeSet) {
  DEBUG printf("in MST Graph1\n");
  N = no_of_nodes;
  DEBUG
  for (int i = 0; i < t; i++)
    printf("T %d\n", terminals[i]);

  // for printing  the parent arrays
  DEBUG
  for (int i = 0; i < t; i++) {
    printf("Parent of %d\n", terminals[i]);
    for (int j = 0; j < N; j++) {
      printf("P[%d]=%d\n", j, h_parentArrays[j + i * N]);
    }
  }

  DEBUG printf("Reading INPUT \n");

  // IMPORTANT CONSTRUCT G' on TE
  no_of_nodes = t;
  edge_list_size = t * (t - 1);

  // allocate host memory
  hostMemAllocationNodes();

  // initalize the memory
  for (int i = 0; i < no_of_nodes; i++) {
    start = i * (t - 1);
    edgeno = t - 1;
    h_graph_nodes[i].starting = start;
    h_graph_nodes[i].no_of_edges = edgeno;
    sameindex[i] = i;
    falseval[i] = false;
    trueval[i] = true;
    infinity[i] = INF;
    zero[i] = 0;
    h_maxid_maxdegree[i] = -1;
  }

  // read the source node from the file, not needed here though
  source = 0;

  DEBUG
  for (int i = 0; i < no_of_nodes; i++)
    printf("V %d: %d,%d\n", i, h_graph_nodes[i].starting, h_graph_nodes[i].no_of_edges);

  DEBUG printf("n=%d 2m=%d \n", no_of_nodes, edge_list_size);

  DEBUG printf("Reading %d edges\n", edge_list_size);

  hostMemAllocationEdges();
  DEBUG printf("BFORE for\n");
  for (int i = 0; i < edge_list_size; i++) {
    h_graph_edges[i] = edges[i];
    h_graph_weights[i] = edges_wt[i];

    h_graph_MST_edges[i] = false;
    DEBUG printf("%d: -- %d: %d\n", i, h_graph_edges[i], h_graph_weights[i]);
  }

  // Copy the Node list to device memory
  deviceMemAllocateNodes();
  deviceMemAllocateEdges();
  deviceMemCopy();

  GPUMST();

  DEBUG printf("MST1 Compleet\n");
  hipMemcpy(test, d_graph_colorindex, sizeof(int) * no_of_nodes, hipMemcpyDeviceToHost);
  for (int i = 0; i < no_of_nodes; i++) {
    if (test[i] != 0) {
      printf("1:All Colors not 0, Error at %d\n", i);
      break;
    }
  }

  int q = 0;
  int minimumCost = 0;
  DEBUG printf("Final edges present in MST\n");
  hipMemcpy(h_graph_MST_edges, d_graph_MST_edges, sizeof(bool) * edge_list_size, hipMemcpyDeviceToHost);

  int v1 = 0;  // IMP to

  for (int i = 0; i < int(edge_list_size); i++) {
    int v1Limit = h_graph_nodes[v1].starting + h_graph_nodes[v1].no_of_edges;
    if (i == v1Limit)  // if limit reached, move to next v1
      v1++;

    //~ printf("%d :",i);
    if (h_graph_MST_edges[i]) {
      int v2 = h_graph_edges[i];
      int edgeweight = h_graph_weights[i];

      vSet.insert(v1);
      vSet.insert(v2);
      if (v1 < v2)
        eSet.insert(make_pair(v1, v2));
      else
        eSet.insert(make_pair(v2, v1));

      minimumCost += edgeweight;
      q++;
    }
    // Post increment after printing!
  }

  DEBUG printf("Printing Parent array\n");

  for (std::set<pair<int, int>>::iterator it = eSet.begin(), end = eSet.end(); it != end; ++it) {
    int v1 = it->first;
    int u = terminals[v1];
    int v2 = it->second;
    int v = terminals[v2];

    //~ printf("%d -- %d\n", u,v);
    PrintParentOf(N, h_parentArrays, v1, u, v, stEdges);  // populates stEdges
  }

  unsigned mstVal = 0;

  for (std::set<pair<int, int>>::iterator it = stEdges.begin(), end = stEdges.end(); it != end; ++it) {
    int v1 = it->first;
    int v2 = it->second;
    mstVal += W[make_pair(v1, v2)];
    nodeSet.insert(v1);
    nodeSet.insert(v2);
  }

  if (stEdges.size() == nodeSet.size() - 1) {  // MST(G') is tree alread then we do not have to do G"
                                               // TIMER STOP
    hipEventRecord(tstop);
    hipEventSynchronize(tstop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, tstart, tstop);
    totalTimeMilliSec += milliseconds;

    if (printEdges) {
      for (std::set<pair<int, int>>::iterator it = stEdges.begin(), end = stEdges.end(); it != end; ++it) {
        int v1 = it->first;
        int v2 = it->second;
        printf("%d %d\n", v1 + 1, v2 + 1);
      }
    }
    printf("VALUE %d,%f\n", mstVal, totalTimeMilliSec);
  }

  //! freeMem(); // This frees the GPU memory as well! OMG!
  DEBUG printf("in MST Graph1\n");
}

// Construct G" and MST(G")
void MSTGraphG2(set<pair<int, int>>& stEdges, set<int>& nodeSet, map<pair<int, int>, int>& W) {
  DEBUG printf("in MST Graph2\n");
  N = nodeSet.size();  // need to be modified.
  vector<vector<int>> graph(N);

  unordered_map<int, int> vMap;
  vector<int> nodeVec(nodeSet.begin(), nodeSet.end());
  int i = 0;
  for (auto& a : nodeVec) {
    vMap[a] = i++;
  }

  for (auto& a : stEdges) {
    int v1 = vMap[a.first];
    int v2 = vMap[a.second];
    graph[v2].push_back(v1);
    graph[v1].push_back(v2);
  }

  //~ printf("In MST\n");
  int eSize = stEdges.size() * 2;

  DEBUG printf("Reading INPUT \n");
  //~ scanf("%d",&no_of_nodes);

  // allocate host memory
  DEBUG printf("Reading %d nodes	", no_of_nodes);

  // IMPORTANT
  no_of_nodes = N;         // n
  edge_list_size = eSize;  // 2m

  hostMemAllocationNodes();
  int cumSum = 0;
  // initalize the memory
  for (int i = 0; i < no_of_nodes; i++) {
    //~ fscanf(fp,"%d %d",&start,&edgeno);
    //~ scanf("%d %d",&start,&edgeno);
    start = cumSum;  // start of csr(i) for i \in V
    auto adjSize = graph[i].size();
    cumSum += adjSize;
    edgeno = adjSize;  // |N(i)|
    h_graph_nodes[i].starting = start;
    h_graph_nodes[i].no_of_edges = edgeno;
    sameindex[i] = i;  // i // this is good!
    falseval[i] = false;
    trueval[i] = true;
    infinity[i] = INF;
    zero[i] = 0;
    h_maxid_maxdegree[i] = -1;
  }

  // read the source node from the file, not needed here though
  //~ scanf("%d",&source);.
  source = 0;

  //~ scanf("%d",&edge_list_size);
  DEBUG
  for (int i = 0; i < no_of_nodes; i++)
    printf("V %d: %d,%d\n", i, h_graph_nodes[i].starting, h_graph_nodes[i].no_of_edges);

  DEBUG printf("n=%d 2m=%d \n", no_of_nodes, edge_list_size);

  DEBUG printf("Reading %d edges\n", edge_list_size);

  //~ int id,cost;

  hostMemAllocationEdges();
  DEBUG printf("BFORE for\n");

  int u = 0;
  i = 0;
  for (auto adjList : graph) {
    int v1 = nodeVec[u];  // Thanks Rupesh!

    for (auto v : adjList) {
      int v2 = nodeVec[v];

      //! printf(" %d %d: %d\n",v1,v2 , W[{v1,v2}]);

      h_graph_edges[i] = v;
      h_graph_weights[i] = W[{v1, v2}];
      h_graph_MST_edges[i] = false;

      ++i;
    }
    ++u;
  }

  //~ DEBUG printf("Finished Reading INPUT\n");
  //~ DEBUG printf("Copying Everything to GPU memory\n");

  //~ Copy the Node list to device memory
  deviceMemAllocateNodes();
  deviceMemAllocateEdges();
  deviceMemCopy();

  GPUMST();

  hipMemcpy(test, d_graph_colorindex, sizeof(int) * no_of_nodes, hipMemcpyDeviceToHost);
  for (int i = 0; i < no_of_nodes; i++) {
    if (test[i] != 0) {
      printf("2:All Colors not 0, Error at %d\n", i);
      break;
    }
  }

  int q = 0;
  int minimumCost = 0;
  DEBUG printf("Final edges present in MST\n");
  hipMemcpy(h_graph_MST_edges, d_graph_MST_edges, sizeof(bool) * edge_list_size, hipMemcpyDeviceToHost);

  // TIMER STOP
  hipEventRecord(tstop);
  hipEventSynchronize(tstop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, tstart, tstop);
  totalTimeMilliSec += milliseconds;

  int v1 = 0;    // IMPORTANT to INIT
  eSet.clear();  // reset!
  for (int i = 0; i < int(edge_list_size); ++i) {
    int v1Limit = h_graph_nodes[v1].starting + h_graph_nodes[v1].no_of_edges;
    if (i == v1Limit)  // if limit reached, move to next v1
      v1++;

    if (h_graph_MST_edges[i]) {
      int v2 = h_graph_edges[i];
      int edgeweight = h_graph_weights[i];

      int u = nodeVec[v1] + 1;  // for printing
      int v = nodeVec[v2] + 1;

      if (printEdges)
        printf("%d %d \n", u, v);

      minimumCost += edgeweight;
      q++;
    }
  }
  // For each terminal on their respective parent array!

  printf("VALUE %d,%f, %f\n", minimumCost, totalTimeMilliSec, milliseconds);

  DEBUG printf("in MST Graph2\n");
  freeMem();
}

////////////////////////////////////////////////////////////////////////////////
// KMBGPU ALGORITHM using CUDA
////////////////////////////////////////////////////////////////////////////////

void KMBAlgo(int argc, char** argv) {
  DEBUG printf("Using sCount:%d\n", sCount);

  // Use if required on multiGPU device
  // hipSetDevice(1);

  size_t mf, ma;
  hipError_t err = hipMemGetInfo(&mf, &ma);
  if (err != hipSuccess)
    printf("ALERT: %s \n", hipGetErrorString(err));

  int* source = (int*)malloc(sizeof(int) * (sCount));

  scanf("%d", &no_of_nodes);
  DEBUG printf("|V|: %d\n", no_of_nodes);

  int num_of_blocks = 1;
  int num_of_threads_per_block = no_of_nodes;

  // Make execution Parameters according to the number of nodes
  // Distribute threads across multiple Blocks if necessary
  if (no_of_nodes > MAX_THREADS_PER_BLOCK) {
    num_of_blocks = (int)ceil(no_of_nodes / (double)MAX_THREADS_PER_BLOCK);
    num_of_threads_per_block = MAX_THREADS_PER_BLOCK;
  }
  // initalize the memory
  // allocate host memory
  int* h_graph_nodes = (int*)malloc(sizeof(int) * (no_of_nodes + 1));  //  +1 for csrM

  int start, edgeno;

  int no = 0;
  for (unsigned int i = 0; i < no_of_nodes; i++) {
    scanf("%d %d", &start, &edgeno);
    DEBUG printf("%d %d\n", start, edgeno);
    if (edgeno > 100)
      no++;
    h_graph_nodes[i] = start;
  }

  h_graph_nodes[no_of_nodes] = start + edgeno;

  // read the source int from the file

  int dummy;
  scanf("%d", &dummy);  // not use else where

  scanf("%d", &edge_list_size);

  int* h_graph_edges = (int*)malloc(sizeof(int) * edge_list_size);
  int* h_graph_weights = (int*)malloc(sizeof(int) * edge_list_size);

  map<pair<int, int>, int> W;

  int id;

  for (int i = 0, j = 0; i < edge_list_size; i++) {
    int id1;
    if (i >= h_graph_nodes[j + 1]) j++;
    scanf("%d %d", &id, &id1);
    h_graph_edges[i] = id;
    h_graph_weights[i] = id1;
    DEBUG printf("%d %d\n", h_graph_edges[i], h_graph_weights[i]);

    W[make_pair(j, id)] = id1;
    W[make_pair(id, j)] = id1;
  }

  int terminalSize;

  scanf("%d", &terminalSize);

  int terminals[terminalSize];

  for (int i = 0; i < terminalSize; i++) {
    scanf("%d", &id);
    terminals[i] = id;
  }

  h_parentArrays = (int*)malloc(sizeof(int) * (no_of_nodes * terminalSize));
  hipMalloc((void**)&d_parentArrays, sizeof(int) * no_of_nodes * terminalSize);

  int edgeId = 0;

  edges = (int*)malloc(sizeof(int) * terminalSize * (terminalSize - 1));  // For MST
  edges_wt = (int*)malloc(sizeof(int) * terminalSize * (terminalSize - 1));

  // setup execution parameters
  dim3 grid(num_of_blocks, 1, 1);
  dim3 threads(num_of_threads_per_block, 1, 1);

  // Copy the int list to device memory
  int* d_graph_nodes;
  hipMalloc((void**)&d_graph_nodes, sizeof(int) * (no_of_nodes + 1));  //+1 for csrM

  // Copy the Edge List to device Memory
  int* d_graph_edges;
  hipMalloc((void**)&d_graph_edges, sizeof(int) * edge_list_size);

  int* d_graph_weights;
  hipMalloc((void**)&d_graph_weights, sizeof(int) * edge_list_size);

  // allocate mem for the result on host side
  int* h_cost = (int*)malloc(sizeof(int) * no_of_nodes * sCount);  // Rupesh

  // allocate device memory for result / OUTPUT
  int* d_cost;
  hipMalloc((void**)&d_cost, sizeof(int) * no_of_nodes * sCount);

  int* h_parent = (int*)malloc(sizeof(int) * no_of_nodes * sCount);

  // copy the parent array
  int* d_parent;
  hipMalloc((void**)&d_parent, sizeof(int) * no_of_nodes * sCount);

  bool* d_changed;
  bool* changed = (bool*)malloc(sizeof(bool));
  hipMalloc((void**)&d_changed, sizeof(bool));
  cudaCheckError();

  // new for kSSSP
  int* d_sources;
  hipMalloc((void**)&d_sources, sizeof(int) * sCount);
  cudaCheckError();

  /*************
   * TIMER START
   *************/
  hipEventCreate(&tstart);
  hipEventCreate(&tstop);
  hipEventRecord(tstart);
  cudaCheckError();

  hipMemcpy(d_graph_nodes, h_graph_nodes, (sizeof(int) * (no_of_nodes + 1)), hipMemcpyHostToDevice);  // +1 for csrM
  cudaCheckError();

  hipMemcpy(d_graph_weights, h_graph_weights, sizeof(int) * edge_list_size, hipMemcpyHostToDevice);
  cudaCheckError();

  hipMemcpy(d_graph_edges, h_graph_edges, sizeof(int) * edge_list_size, hipMemcpyHostToDevice);
  cudaCheckError();

  int tempScount = sCount;  // Just to ensure the last run runs < sCount times
  printf("sCount:%d terminalSize:%d n:%d m:%d\n", sCount, terminalSize, no_of_nodes, edge_list_size);

  for (int it = 0, end = (terminalSize + sCount - 1) / sCount; it < end; ++it) {  // ceil(terminalSize/sCount)

    if (terminalSize % sCount == 0 || it != end - 1) {  // Thanks Rupesh. termSize%sCount==0 || it!=end-1
      tempScount = sCount;
      for (int ii = 0; ii < sCount; ++ii) {
        source[ii] = terminals[sCount * it + ii];
        DEBUG printf("\t #%d SSSP from %d\n", ii + 1, source[ii]);
      }
      num_of_blocks = (sCount * no_of_nodes + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;

    } else {
      tempScount = terminalSize % sCount;  // suposedly 1 t0 sCount-1 for the last round when sCount does not divide terminalSize
      for (int ii = 0, endII = tempScount; ii < endII; ++ii) {
        source[ii] = terminals[sCount * it + ii];
        DEBUG printf("\t #%d SSSP from %d\n", ii + 1, source[ii]);
      }

      num_of_blocks = (tempScount * no_of_nodes + MAX_THREADS_PER_BLOCK - 1) / MAX_THREADS_PER_BLOCK;
    }

    num_of_threads_per_block = MAX_THREADS_PER_BLOCK;  //(no_of_nodes<MAX_THREADS_PER_BLOCK? no_of_nodes: MAX_THREADS_PER_BLOCK);
    dim3 gridKN(num_of_blocks, 1, 1);
    dim3 threadsKN(num_of_threads_per_block, 1, 1);

    int k = 0;

    hipMemcpy(d_sources, source, (sizeof(int) * (tempScount)), hipMemcpyHostToDevice);        //
    kernelInitDistAndParent<<<gridKN, threadsKN>>>(no_of_nodes, d_cost, d_parent, tempScount);  // SAME grid
    kernelInitSources<<<1, 1>>>(no_of_nodes, d_sources, d_cost, tempScount);                    //

    cudaCheckError();

    do {
      changed[0] = false;

      hipMemcpy(d_changed, changed, sizeof(bool), hipMemcpyHostToDevice);

      cudaCheckError();

      csrKernelBellmanFordMoore<<<gridKN, threadsKN>>>(no_of_nodes, d_sources,
                                                       d_graph_nodes, d_graph_edges, d_graph_weights,  // inputs
                                                       d_changed,                                      // fixed pt var
                                                       d_cost, d_parent,                               // these are outputs
                                                       sCount,
                                                       tempScount);

      cudaCheckError();

      hipMemcpy(changed, d_changed, sizeof(bool), hipMemcpyDeviceToHost);
      cudaCheckError();

      k++;

      DEBUG printf("%d -- FINSHED? %s\n", k, (!changed[0] ? "Yes" : "No"));
    } while (changed[0] == true);
    DEBUG printf("AFTER LAUNCH\n");

    DEBUG printf("\nTOTAL IT:%d\n", k);

    cpyParentArrayNew<<<gridKN, threadsKN>>>(no_of_nodes, it, d_parentArrays, d_parent, sCount, tempScount);  //~DOUBLE~  K COPY
    cudaCheckError();

    // copy result from device to host
    hipMemcpy(h_cost, d_cost, sizeof(int) * no_of_nodes * tempScount, hipMemcpyDeviceToHost);  //~DOUBLE~  K COPY

    unsigned long long int sol;

    DEBUG printf("N=%d %d\n", no_of_nodes, INT_MAX / 2);
    DEBUG
    for (int jj = 0; jj < tempScount; ++jj) {
      sol = 0;
      for (int i = 0; i < no_of_nodes; ++i) {
        sol += h_cost[jj * no_of_nodes + i];
      }
      /// for debugging
      if (printHash) printf("iterat:%d SSSP %d on src %d HASH VAL %lld\n", it, jj + 1, source[jj], sol);
    }

    for (int ii = 0; ii < tempScount; ++ii) {
      int pt1 = terminals[sCount * it + ii];
      for (int j = 0; j < terminalSize; ++j) {
        int pt2 = terminals[j];
        if (pt1 != pt2) {
          edges[edgeId] = j;
          edges_wt[edgeId] = h_cost[no_of_nodes * ii + pt2];  /// Mod
          edgeId++;
        }
      }
    }
  }

  hipMemcpy(h_parentArrays, d_parentArrays, sizeof(int) * no_of_nodes * terminalSize, hipMemcpyDeviceToHost);  // why is this needed? It is used inside MST1

  DEBUG printf("GPU [ms]:%f\n", totalTimeMilliSec);

  // Construct G' and Launch the kernel for the MST(G')
  set<pair<int, int>> stEdges;
  set<int> nodeSet;
  DEBUG printf("In main before MST\n");
  MSTGraph(terminalSize, terminals, W, stEdges, nodeSet);

  // Construct G" and Launch the kernel for the MST(G")
  if (stEdges.size() != nodeSet.size() - 1)  //|E| != |V|-1
    MSTGraphG2(stEdges, nodeSet, W);

  free(h_graph_nodes);
  free(h_graph_edges);
  free(h_graph_weights);
  free(h_cost);

  hipFree(d_graph_nodes);
  hipFree(d_graph_edges);

  hipFree(d_graph_weights);

  hipFree(d_cost);
  hipFree(d_parentArrays);
}
